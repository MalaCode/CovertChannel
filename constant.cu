#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include "repeat.h"
#include "support.h"
#define CARRAY_SIZE 512  //2kb divided by 4

__constant__ unsigned int d_carray[CARRAY_SIZE];
__constant__ unsigned int d_carray1[CARRAY_SIZE];

unsigned int h_test[CARRAY_SIZE];
__device__ uint get_smid(void) {

       uint ret;
       asm("mov.u32 %0, %smid;" : "=r"(ret) );
       return ret;
}

__global__ void Spy (unsigned int *ts, unsigned int *out, int p1, int p2, int its2)
{//ts is the allocate memory on GPU, out is the output, p1 is 1, p2 is 3, its2 is 20
    int t1 = p1; int t2 = p1*p1; int t3 = p1*p1+p1; int t4 = p1*p1+p2; //t1 = 1, t2 = 1, t3 = 2, t4 = 4
    int start_time, end_time;

    unsigned int p;
    int p_start = 0;     
    p = p_start;
    start_time = clock(); //starts clock
                   
    for (int i=0;i<its2;i++) //occurs 20 times
    {
        repeat16(p = d_carray[p];) //repeats this 16x20 times, each time setting p = to an element in d_carray which contains the location-cache values, so p travels down the cache lines reading each value
				   //how does this read from the cache? is d_carray placed into the cache? I was under the impression that TS what the allocated cache memory on the GPU
    }
    
    end_time = clock(); //ends clock
    t1 = p; //sets t1 to end of the cache
    out[0] = t1+t2+t3+t4; //output is set to 4 + end of cache? why?
    if (threadIdx.x == 0)
    {
        ts[blockIdx.x * 2] = start_time; //Sets the 0th entry on the cache line to start time, also is this the handshake?
        ts[blockIdx.x * 2 + 1] = end_time; //Sets the 1st entry on the cache line to start time, also is this the handshake?
    }
//if(threadIdx.x==0)
//printf("1111 block %d smid %d\n", blockIdx.x, get_smid());
}


__global__ void Trojan (int k, unsigned int *ts, unsigned int *out, int p1, int p2, int its2)
{
  if (k %3 ==0 ){ //skips every third iteration, (why?)
  }
  else {
      int t1 = p1; int t2 = p1*p1; int t3 = p1*p1+p1; int t4 = p1*p1+p2; //t1 = 1, t2 = 1, t3 = 2, t5 = 4
      int start_time, end_time;

      unsigned int p;
      int p_start = 16;    //different starting place then spy,why?
      p = p_start;
      start_time = clock();
      for (int i=0;i<its2;i++)
      {
          repeat16(p = d_carray1[p];) //repeats 16x20 times, each time setting p equal to the next stride, however this starts at the 16th index of the array and travles from there, 
		  		      //how does this place values into the cache?
      }
      end_time = clock();		//end time
      t1 = p;				//t1 is end of the cache
      out[0] = t1+t2+t3+t4;		//output is set to 4 + end of cache? why?
  }

//if(threadIdx.x==0)
//printf("2222 block %d smid %d\n", blockIdx.x, get_smid());
}
void cmem_stride(unsigned int *h_carray,unsigned int *h_carray1, unsigned int *d_ts, unsigned int *d_out, unsigned int *ts, int stride, int min_size, int max_size, int step_size)
{   //h_carray is the spy allocation, h_carray1 is the trojan allocation, d_ts is the address of allocated memeory, d_out is where output will be, ts is output from kernel,
    //stride is num cacheset/size cache line, min_size is ?, max_size is ?, step_size is ?
    dim3 Db = dim3(256);	//why 256?, Creates a grid of size 256 x 1 x 1
    dim3 Dg = dim3(10,1,1);     //first param is the number of SMs, created a grid of 10 x 1 x 1
    dim3 Db1 = dim3(256);	//why 256?, creates a grid of size 256 x 1 x 1
    dim3 Dg1 = dim3(10,1,1);	//first param is the number of SMs, creates a grid of 10 x 1 x 1
    hipStream_t stream3, stream1,stream2; //Creates stream in order to run concurrently
    hipStreamCreate(&stream3);	//Create stream3
    hipStreamCreate(&stream1); //create steam1
    hipStreamCreate(&stream2); //create stream2
    hipError_t errcode;
    Timer timer;
    int size = 512;		//Why is this constant? shouldn't it be the number of sets?
    printf ("Constant memory, %d-byte stride\n", stride*4);   //number of cache sets
    printf ("  [array size]: [clocks per read], [max], [min]\n");
    
    // Set up array contents
    for (int i=0;i<size;i++)
    {
        h_carray[i] = i+stride;  //placing values into the array, values are used to make sure that the values is always on the same cache line
        if (h_carray[i] >= size) //check to see if the value exceeds the size of the array, so that we can move to the next cache line
        h_carray[i] %= stride;	 //moding by the stride allows to to maintain the current cache line
    }
    for (int i=0;i<size;i++)
    {
        h_carray1[i] = i+stride; //Same as above
        if (h_carray1[i] >= size)  //Same as above
        h_carray1[i]= (h_carray1[i] % stride); //Same as above
    }
    hipMemcpyToSymbol(HIP_SYMBOL(d_carray), h_carray, CARRAY_SIZE*4); //copies CARRAY_SIZE*4 bytes of h_carray into d_carray
    hipMemcpyToSymbol(HIP_SYMBOL(d_carray1), h_carray1, CARRAY_SIZE*4); //copies CARRAY_SIZE*4 bytes of h_carray1 into d_carray1
    unsigned long long sum_time[14] ;
    
    for(int j=0; j < 14; j++){		//empties the array used to hold the sum_times
      sum_time [j] = 0;			
    }	
    unsigned int max_time[14], min_time[14];     //=(unsigned)-1;
    for(int j=0; j < 14; j++){
      max_time [j] = 0;
    }
    for(int j=0; j < 14; j++){
    min_time [j] =(unsigned)-1;
    }

    int kits = 50;    	//Why this value?
    int its = 20;	//Why this value?

    for (int k = 0; k < kits; k++) //runs 50 times
    {
        startTime(&timer); //starts time
        // Launch kernel

        Spy<<<Dg, Db, 0, stream1>>> (d_ts, d_out, 1,3, its); //launches 256 blocks with 10 threads each with a shared memory of 0, spy is passed the allocated memory, memory for output, 1 (?), 3(?), and its
        Trojan<<<Dg1, Db1, 0, stream2>>> (k, d_ts, d_out, 1,3, its); //launches 256 blocks with 10 threads each with a shared memory of 0, trojan is passed k, allocated memory, memory for output, 1(?), 2(), its
       
        hipDeviceSynchronize();				//makes sure that the trojan and spy are done befor emoving to next iteration

        stopTime(&timer); printf("eennnnddd %f s\n", elapsedTime(timer)); // ends the timer for both spy and trojan
        hipMemcpy(ts, d_ts, 640, hipMemcpyDeviceToHost);		//copies 640 bytes of d_ts to ts
        for(int j=0; j < 10; j++){
    //        sum_time[j] += ts[2*j+1]-ts[2*j];
  //          if (ts[2*j+1]-ts[2*j] > max_time[j]) max_time[j] = ts[2*j+1]-ts[2*j];
//            if (ts[2*j+1]-ts[2*j] < min_time[j]) min_time[j] = ts[2*j+1]-ts[2*j];
            printf (" k = %d: latency %.3f clk\n", k,(ts[2*j+1]-ts[2*j])/(its * 16.0)); 
        }

    }
    /*for(int i=0; i < 14; i++){
    printf ("  %d: %.3f, %.3f, %.3f clk\n", size*4, 
    sum_time[i]/(kits*its*4.0),
    min_time[i]/(its*4.0),
    max_time[i]/(its*4.0));
    }*/
    printf ("\n");
}


int main()
{
    unsigned int ts[4096];  // ts, output from kernel. Two elements used per thread.
    unsigned int *d_ts;	   //used to hold the address of the allocated memory on device
    unsigned int *d_out;  // Unused memory for storing output
    unsigned int *h_carray; //memory allocation for spy
    unsigned int *h_carray1; //memory allocation for trojan

 int nDevices;
         hipGetDeviceCount(&nDevices);
	         for (int i = 0; i < nDevices; i++) {
			           hipDeviceProp_t prop;
				             hipGetDeviceProperties(&prop, i);
					                    //hipSetDevice(1);
							              printf("Device Number: %d\n");
								                printf("  Device name: %s\n", prop.name);
									                  printf("  total Global memory (in bytes): %d\n",
												                    prop.totalGlobalMem);
										           printf("  total shared memory per block (in bytes): %d\n",
												                   prop.sharedMemPerBlock);
										           printf("  Register per block: %d\n",
												                  prop.regsPerBlock);
										           printf(" warp size: %d\n",
												                  prop.warpSize);
										           printf(" maxThreadsPerBlock: %d\n",
												                  prop.maxThreadsPerBlock);
										           printf(" maxThreadsDim1: %d\n",
												                  prop.maxThreadsDim[0]);
										           printf(" maxThreadsDim2: %d\n",
												                  prop.maxThreadsDim[1]);
										           printf(" maxThreadsDim3: %d\n",
												                  prop.maxThreadsDim[2]);
										           printf(" maxGridsize1: %d\n",
												                  prop.maxGridSize[0]);
										           printf(" maxGridsize2: %d\n",
												                  prop.maxGridSize[1]);
										           printf(" maxGridsize3: %d\n",
												                  prop.maxGridSize[2]);
										           printf(" clockrate: %d\n",
												                  prop.clockRate);
										           printf(" const memory: %d\n",
												                  prop.totalConstMem);
										           printf(" device overlap: %d\n",
												                  prop.deviceOverlap);
										           printf("multiProcessorCount : %d\n",
												                  prop.multiProcessorCount);
										           printf("integrated : %d\n",
												                  prop.integrated);
										
										           printf("concurrentKernels: %d\n",
												                  prop.concurrentKernels);
										           printf("asyncEngineCount: %d\n",
												                  prop.asyncEngineCount);
										           printf("maxThreadPerMultiProcessor: %d\n",
												                  prop.maxThreadsPerMultiProcessor);
										           printf(" L2 cache size (in bytes): %d\n",
												                  prop.l2CacheSize);
										           printf(" minor: %d\n",
												                  prop.minor);
										           printf(" major: %d\n", 
														                     prop.major);


													           }
														 
																																												       		  // Allocate device array.
    hipError_t errcode;
    if (hipSuccess != (errcode = hipMalloc((void**)&d_ts, sizeof(ts)))) //attempting to allocate memory on GPU
    {
        printf ("hipMalloc failed %s:%d\n", __FILE__, __LINE__);
        printf ("   %s\n", hipGetErrorString(errcode));
        return -1;
    }
    if (hipSuccess != hipMalloc((void**)&d_out, 4))			  //attempting to allocate 4 bytes of memory, (why?)
    {
        printf ("hipMalloc failed %s:%d\n", __FILE__, __LINE__);
        return -1;
    }
    h_carray = (unsigned int*)malloc(CARRAY_SIZE*4);  //Size of the cache line * size of cache set
    h_carray1 = (unsigned int*)malloc(CARRAY_SIZE*4); //Size of the cache line * size of cache set


    // Stride 16 L1
    cmem_stride(h_carray,h_carray1, d_ts, d_out, ts, 512/4, 512-64, 2048+192, 16/4); //first num is the number of cache sets divded by the size of each cache line

    //free memory
    hipFree(d_ts);
    hipFree(d_out);
    free(h_carray);
        free(h_carray1);
    return 0;
}
void startTime(Timer* timer) {
    gettimeofday(&(timer->startTime), NULL);
}

void stopTime(Timer* timer) {
    gettimeofday(&(timer->endTime), NULL);
}

float elapsedTime(Timer timer) {
    return ((float) ((timer.endTime.tv_sec - timer.startTime.tv_sec) \
                + (timer.endTime.tv_usec - timer.startTime.tv_usec)/1.0e6));
}

